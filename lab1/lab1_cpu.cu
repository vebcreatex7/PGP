
#include <hip/hip_runtime.h>
#include <iostream>
#include "stdio.h"
#include <stdlib.h>
#include <iomanip>


void add(double *a, double *b, double *c, size_t n) {
    for (int i = 0; i != n; i++) {
        c[i] = a[i] + b[i];
    }
}


int main() {
    size_t n;
    scanf("%ld", &n);

    double *a, *b, *c;
    a = (double*)malloc(sizeof(double) * n);
    b = (double*)malloc(sizeof(double) * n);
    c = (double*)malloc(sizeof(double) * n);

    for (size_t i = 0; i != n; i++)
        scanf("%lf", &a[i]);
    for (size_t i = 0; i != n; i++)
        scanf("%lf", &b[i]);

    
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    add(a, b, c, n);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(stop);
    hipEventDestroy(start);
    printf("%f\n", time);

    /*
    for (size_t i = 0; i != n; i++)
        printf("%10.10e", c[i]);
    printf("\n");
    */
    
}