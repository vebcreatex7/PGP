
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "stdlib.h"
#include <chrono>


__global__ void add(double *a, double *b, double *sum, size_t n) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    while (idx < n) {
        sum[idx] = a[idx] + b[idx];
        idx += blockDim.x * gridDim.x;
    }
}


int main() {

    size_t Blocks, Threads;
    scanf("%ld %ld", &Blocks, &Threads);

    size_t n;
    scanf("%ld", &n);

    double *a, *b, *c;
    a = (double*)malloc(sizeof(double) * n);
    b = (double*)malloc(sizeof(double) * n);
    c = (double*)malloc(sizeof(double) * n);

    double *dev_a, *dev_b, *dev_c;
    hipMalloc((void**)&dev_a, sizeof(double) * n);
    hipMalloc((void**)&dev_b, sizeof(double) * n);
    hipMalloc((void**)&dev_c, sizeof(double) * n);


    for (size_t i = 0; i != n; i++)
        scanf("%lf", &a[i]);
    hipMemcpy(dev_a, a, sizeof(double) * n, hipMemcpyHostToDevice);

    for (size_t i = 0; i != n; i++)
        scanf("%lf", &b[i]);
    hipMemcpy(dev_b, b, sizeof(double) * n, hipMemcpyHostToDevice);

    

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    add<<<Blocks, Threads>>>(dev_a, dev_b, dev_c, n);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("%f\n", time);
    hipMemcpy(c, dev_c, sizeof (double) * n, hipMemcpyDeviceToHost);

    /*
    for (size_t i = 0; i != n; i++)
        printf("%10.10e ", c[i]);
    printf("\n");
    */
    free(a);
    free(b);
    free(c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}