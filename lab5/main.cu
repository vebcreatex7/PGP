#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdlib.h"
#include <algorithm>
#include <iostream>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#define N (1 << 24)

#define BLOCKS 1024
#define THREADS 1024

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

__global__ void histogram(uint* hist, const uint* data, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offsetx = gridDim.x * blockDim.x;

    for (int i = idx; i < size; i += offsetx)
        atomicAdd(hist + data[i], 1);
}

__global__ void restore(const uint* input, uint* output, uint* pref, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int offsetx = blockDim.x * gridDim.x;

    for (int i = idx; i < size; i += offsetx)
        output[atomicAdd(pref + input[i], -1) - 1] = input[i];
}

void CountingSort(const uint *input, uint *output, int size) {
    uint *d_input, *d_output;
    CSC(hipMalloc((void**)&d_input, size * sizeof(uint)));
    CSC(hipMemcpy(d_input, input, size * sizeof(uint), hipMemcpyHostToDevice));
    CSC(hipMalloc((void**)&d_output, size * sizeof(uint)));

    uint* hist;
    CSC(hipMalloc((void**)&hist, N * sizeof(uint)));
    CSC(hipMemset(hist, 0, N * sizeof(uint)));
    
    histogram<<<BLOCKS, THREADS>>>(hist, d_input, size);
    CSC(hipGetLastError());
    CSC(hipDeviceSynchronize());

    thrust::inclusive_scan(thrust::device, hist, hist + N, hist);
    
    restore<<<BLOCKS, THREADS>>>(d_input, d_output, hist, size);
    CSC(hipGetLastError());

    
    CSC(hipMemcpy(output, d_output, sizeof(uint) * size, hipMemcpyDeviceToHost));

    CSC(hipFree(hist));
    CSC(hipFree(d_input));
    CSC(hipFree(d_output));
} 

int main() {
    int n;
    fread(&n, sizeof(int), 1, stdin);

    uint* input = (uint*)malloc(n * sizeof(uint));
    uint* output = (uint*)malloc(n * sizeof(uint));
    fread(input, sizeof(uint), n, stdin);
    
    CountingSort(input, output, n);
    fwrite(output, sizeof(uint), n, stdout);

    free(input);
    free(output);

}
